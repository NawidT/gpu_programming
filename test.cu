

#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void helloCUDA() {
    printf("Hello from CUDA kernel cuda!\n");
}

int main() {
    printf("Hello from CUDA kernel cuda!\n");
    helloCUDA<<<1,1>>>();
    hipDeviceSynchronize();
    printf("Hello from CUDA kernel cuda!\n");
    return 0;
}
